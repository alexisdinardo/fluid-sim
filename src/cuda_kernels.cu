#include "hip/hip_runtime.h"
#include "cuda_kernels.h"
#include <stdio.h>

// Device memory pointers
CUDAParticle* d_particles = nullptr;
int* d_neighbors = nullptr;
int* d_neighborCounts = nullptr;
int maxNeighbors = 64;  // Maximum number of neighbors per particle

// Initialize CUDA
void initCUDA(int maxParticles) {
    hipMalloc(&d_particles, maxParticles * sizeof(CUDAParticle));
    hipMalloc(&d_neighbors, maxParticles * maxNeighbors * sizeof(int));
    hipMalloc(&d_neighborCounts, maxParticles * sizeof(int));
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA initialization error: %s\n", hipGetErrorString(err));
    }
}

// Clean up CUDA memory
void cleanupCUDA() {
    if (d_particles) hipFree(d_particles);
    if (d_neighbors) hipFree(d_neighbors);
    if (d_neighborCounts) hipFree(d_neighborCounts);
    
    d_particles = nullptr;
    d_neighbors = nullptr;
    d_neighborCounts = nullptr;
}

// Upload particles to GPU
void updateCUDAParticles(CUDAParticle* hostParticles, int numParticles) {
    hipMemcpy(d_particles, hostParticles, numParticles * sizeof(CUDAParticle), hipMemcpyHostToDevice);
}

// Download particles from GPU
void downloadCUDAParticles(CUDAParticle* hostParticles, int numParticles) {
    hipMemcpy(hostParticles, d_particles, numParticles * sizeof(CUDAParticle), hipMemcpyDeviceToHost);
}

// CUDA kernel implementations for the physics functions
// These are simple placeholder implementations - you would need to expand these
// with the actual SPH algorithm logic

__global__ void findNeighborsKernel(CUDAParticle* particles, int* neighbors, int* neighborCounts, 
                                    int numParticles, float smoothingRadius) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= numParticles) return;
    
    int count = 0;
    for (int j = 0; j < numParticles; j++) {
        if (i == j) continue;
        
        float2 diff;
        diff.x = particles[i].position.x - particles[j].position.x;
        diff.y = particles[i].position.y - particles[j].position.y;
        
        float distSq = diff.x * diff.x + diff.y * diff.y;
        
        if (distSq < smoothingRadius * smoothingRadius && count < 64) {
            neighbors[i * 64 + count] = j;
            count++;
        }
    }
    
    neighborCounts[i] = count;
}

__global__ void calculateDensityPressureKernel(CUDAParticle* particles, int* neighbors, int* neighborCounts,
                                             int numParticles, float smoothingRadius, 
                                             float particleMass, float restDensity, float pressureConstant) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= numParticles) return;
    
    // Simple density calculation
    float density = 0.0f;
    int count = neighborCounts[i];
    
    for (int j = 0; j < count; j++) {
        int neighborIdx = neighbors[i * 64 + j];
        
        float2 diff;
        diff.x = particles[i].position.x - particles[neighborIdx].position.x;
        diff.y = particles[i].position.y - particles[neighborIdx].position.y;
        
        float distSq = diff.x * diff.x + diff.y * diff.y;
        float dist = sqrtf(distSq);
        
        if (dist < smoothingRadius) {
            // Simple poly6 kernel
            float term = 1.0f - (dist * dist) / (smoothingRadius * smoothingRadius);
            density += particleMass * term * term * term;
        }
    }
    
    particles[i].density = density;
    
    // Calculate pressure using equation of state
    particles[i].pressure = pressureConstant * (density - restDensity);
    if (particles[i].pressure < 0.0f) particles[i].pressure = 0.0f;
}

__global__ void calculateForcesKernel(CUDAParticle* particles, int* neighbors, int* neighborCounts,
                                    int numParticles, float smoothingRadius, float viscosity, float gravity) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= numParticles) return;
    
    float2 force;
    force.x = 0.0f;
    force.y = -gravity;  // Apply gravity
    
    int count = neighborCounts[i];
    
    for (int j = 0; j < count; j++) {
        int neighborIdx = neighbors[i * 64 + j];
        
        float2 diff;
        diff.x = particles[i].position.x - particles[neighborIdx].position.x;
        diff.y = particles[i].position.y - particles[neighborIdx].position.y;
        
        float distSq = diff.x * diff.x + diff.y * diff.y;
        float dist = sqrtf(distSq);
        
        if (dist > 0.0001f && dist < smoothingRadius) {
            // Simple pressure and viscosity force calculation
            float2 dir;
            dir.x = diff.x / dist;
            dir.y = diff.y / dist;
            
            // Basic pressure force
            float pressureGradient = (particles[i].pressure + particles[neighborIdx].pressure) / 
                                     (2.0f * particles[neighborIdx].density);
            
            float pressureTerm = pressureGradient * (1.0f - dist / smoothingRadius) * (1.0f - dist / smoothingRadius);
            
            force.x += dir.x * pressureTerm;
            force.y += dir.y * pressureTerm;
            
            // Basic viscosity force
            float2 velDiff;
            velDiff.x = particles[neighborIdx].velocity.x - particles[i].velocity.x;
            velDiff.y = particles[neighborIdx].velocity.y - particles[i].velocity.y;
            
            float viscosityTerm = viscosity * (1.0f - dist / smoothingRadius) / particles[neighborIdx].density;
            
            force.x += velDiff.x * viscosityTerm;
            force.y += velDiff.y * viscosityTerm;
        }
    }
    
    // Limit force magnitude to prevent explosion
    float forceSq = force.x * force.x + force.y * force.y;
    float maxForce = 100.0f;
    
    if (forceSq > maxForce * maxForce) {
        float scale = maxForce / sqrtf(forceSq);
        force.x *= scale;
        force.y *= scale;
    }
    
    particles[i].force = force;
}

__global__ void integrateKernel(CUDAParticle* particles, int numParticles, float deltaTime, 
                             float width, float height, float restitution) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= numParticles) return;
    
    // Calculate acceleration
    float2 acceleration;
    acceleration.x = particles[i].force.x / fmaxf(particles[i].density, 0.0001f);
    acceleration.y = particles[i].force.y / fmaxf(particles[i].density, 0.0001f);
    
    // Update velocity
    particles[i].velocity.x += acceleration.x * deltaTime;
    particles[i].velocity.y += acceleration.y * deltaTime;
    
    // Apply damping
    particles[i].velocity.x *= 0.998f;
    particles[i].velocity.y *= 0.998f;
    
    // Update position
    particles[i].position.x += particles[i].velocity.x * deltaTime;
    particles[i].position.y += particles[i].velocity.y * deltaTime;
    
    // Handle boundaries
    float radius = particles[i].radius;
    
    if (particles[i].position.x < radius) {
        particles[i].position.x = radius;
        particles[i].velocity.x *= -restitution;
    } else if (particles[i].position.x > width - radius) {
        particles[i].position.x = width - radius;
        particles[i].velocity.x *= -restitution;
    }
    
    if (particles[i].position.y < radius) {
        particles[i].position.y = radius;
        particles[i].velocity.y *= -restitution;
    } else if (particles[i].position.y > height - radius) {
        particles[i].position.y = height - radius;
        particles[i].velocity.y *= -restitution;
    }
}

__global__ void applyMouseForceKernel(CUDAParticle* particles, int numParticles, 
                                    float2 mousePos, bool mousePressed, 
                                    float mouseRadius, float mouseForce) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= numParticles || !mousePressed) return;
    
    float2 diff;
    diff.x = particles[i].position.x - mousePos.x;
    diff.y = particles[i].position.y - mousePos.y;
    
    float distSq = diff.x * diff.x + diff.y * diff.y;
    float dist = sqrtf(distSq);
    
    if (dist < mouseRadius) {
        // Calculate direction from mouse to particle
        float2 dir;
        if (dist > 0.0001f) {
            dir.x = diff.x / dist;
            dir.y = diff.y / dist;
        } else {
            dir.x = 0.0f;
            dir.y = 1.0f;
        }
        
        // Force strength decreases with distance
        float strength = (1.0f - dist / mouseRadius) * mouseForce;
        
        // Apply force
        particles[i].velocity.x += dir.x * strength;
        particles[i].velocity.y += dir.y * strength;
    }
}

// Wrapper functions to launch the CUDA kernels
void findNeighborsCUDA(int numParticles, float smoothingRadius) {
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;
    
    findNeighborsKernel<<<numBlocks, blockSize>>>(d_particles, d_neighbors, d_neighborCounts, 
                                                numParticles, smoothingRadius);
    hipDeviceSynchronize();
}

void calculateDensityPressureCUDA(int numParticles, float smoothingRadius, 
                               float particleMass, float restDensity, 
                               float pressureConstant) {
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;
    
    calculateDensityPressureKernel<<<numBlocks, blockSize>>>(d_particles, d_neighbors, d_neighborCounts,
                                                         numParticles, smoothingRadius, 
                                                         particleMass, restDensity, pressureConstant);
    hipDeviceSynchronize();
}

void calculateForcesCUDA(int numParticles, float smoothingRadius, 
                      float viscosity, float gravity) {
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;
    
    calculateForcesKernel<<<numBlocks, blockSize>>>(d_particles, d_neighbors, d_neighborCounts,
                                                numParticles, smoothingRadius, viscosity, gravity);
    hipDeviceSynchronize();
}

void integrateCUDA(int numParticles, float deltaTime, 
                float width, float height, float restitution) {
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;
    
    integrateKernel<<<numBlocks, blockSize>>>(d_particles, numParticles, deltaTime, 
                                           width, height, restitution);
    hipDeviceSynchronize();
}

void applyMouseForceCUDA(int numParticles, float2 mousePos, 
                      bool mousePressed, float mouseRadius, float mouseForce) {
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;
    
    applyMouseForceKernel<<<numBlocks, blockSize>>>(d_particles, numParticles, 
                                                mousePos, mousePressed, 
                                                mouseRadius, mouseForce);
    hipDeviceSynchronize();
}